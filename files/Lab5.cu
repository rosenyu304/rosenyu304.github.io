/*
    Author: Ting-Ying Yu
    Class: ECE4122
    Last Date Modified: Nov 21, 2021

    Description:
        2D Steady State Heat Conduction in a Thin Plate

        A CUDA program that determine the steady state heat distribution
        in a thin metal plate using synchronous iteration on a GPU using 
        Laplace's equation finite difference method.
        You will be solving Laplace's
    
    Code References:
        1) https://stackoverflow.com/questions/7876624/timing-cuda-operations
        2) https://github.com/NVIDIA/cuda-samples/blob/master/Samples/vectorAdd/vectorAdd.cu
        3) https://cpp.hotexamples.com/examples/-/-/cudaGetDeviceProperties/cpp-cudagetdeviceproperties-function-examples.html

*/

#include <iostream>
#include <fstream>
#include <string>
#include <cstring>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iomanip>      // std::setprecision
//#include <helper_cuda.h>
#include <iostream>
#include <memory>
#include <sstream>
#include <limits>
#include <climits>
#include <cstddef>
#include <cmath>
#include <stdlib.h> 
#include <ctype.h>
#include <unistd.h>
#include <assert.h>

#include <algorithm>

using namespace std;
bool is_numeric(char* & input, int & nInputNumber);

/*
 * Function: __global__ void UpdatingTemp(double* H, double* G, int N, int totalElements)
 * Description : Calculating the updated temperature by averaging the four neighboring mesh points
 */
__global__ void UpdatingTemp(double* H, double* G, int N, int totalElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if ( (i > N - 1) && (i < totalElements - N) && (i % N != 0) && ((i + 1) % N != 0) )
    {
	    G[i] = 0.25 * ( H[i + 1] + H[i - 1] + H[i + N] + H[i - N] );
    }
}

/*
 * Function: __global__ void copyArray(double* H, double* G, int totalElements)
 * Description : Copy the array with updated temperature to the original array that stores temperatures
 */
__global__ void copyArray(double* H, double* G, int totalElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < totalElements)
    {
	    H[i] = G[i];
    }
}

int main(int argc, char* argv[])
{
    // Open file
    ofstream outputFile("finalTemperatures.csv");

    int N = 0;
    int I = 0;
    int opt = 0;
    bool hasN = false;
    bool hasI = false;
    bool isNValid = false;
    bool isIValid = false;
    
    // If there is clearly no enough inputs
    if (argc < 4)
    {
        outputFile << "Invalid parameters, please check your values.\n";
        cout << "Invalid parameters, please check your values.\n";
        return -1;
    }


    // Check if all inputs are valid
    while ((opt = getopt (argc, argv, "N:I:")) != -1)
    {
        switch (opt)
        {
            case 'N':
                isNValid = is_numeric(optarg, N);

                if (N < 0 || !isNValid)
                {
                    outputFile << "Invalid parameters, please check your values.\n";
                    cout << "Invalid parameters, please check your values.\n";
                    return 0;
                }

                hasN = true;
                break;
            case 'I':
                isIValid = is_numeric(optarg, I);

                if (I < 0 || !isIValid)
                {
                    outputFile << "Invalid parameters, please check your values.\n";
                    cout << "Invalid parameters, please check your values.\n";
                    return 0;
                }
                hasI = true;
                break;

            default:
                outputFile << "Invalid parameters, please check your values.\n";
                cout << "Invalid parameters, please check your values.\n";
                return 0;
        }
        
    }
    
    if (!hasN || !hasI)
    {
        outputFile << "Invalid parameters, please check your values.\n";
        cout << "Invalid parameters, please check your values.\n";
        return 0;
    }

    hipError_t err = hipSuccess;

    // (0) Get plate dimensions
    int plateSideLength = N + 2;
    int totalElements = plateSideLength * plateSideLength;
    size_t size = totalElements * sizeof(double);

    // (1) Allocate CPU memory for the arrays
    double* h_H = (double*)malloc(size);
    double* h_G = (double*)malloc(size);

    if (h_H == NULL || h_G == NULL)
    {
        cout << "Memory allocations for Host failed" << endl;
        exit(EXIT_FAILURE);
    }

    // (2) Initialize the values for the arrays
    for (int i = 0; i < plateSideLength; i++)
    {
        for (int j = 0; j < plateSideLength; j++)
        {
            if ((i == 0) && (j > (0.3 * (plateSideLength - 1)) && (j < 0.7 * (plateSideLength - 1))))
            {
                // 100 degree condition
                h_H[i * plateSideLength + j] = 100.0;
            }
            else
            {
                h_H[i * plateSideLength + j] = 20.0;
            }
        }
    }

    // (3) Allocate GPU memory for array
    double* d_H = (double*)malloc(size);
    err = hipMalloc((void**)&d_H, size);
    if (err != hipSuccess)
    {
        cout << "Memory allocations for GPU failed" << endl;
        exit(EXIT_FAILURE);
    }
    double* d_G = (double*)malloc(size);
    err = hipMalloc((void**)&d_G, size);
    if (err != hipSuccess)
    {
        cout << "Memory allocations for GPU failed" << endl;
        exit(EXIT_FAILURE);
    }

    // (4) Send CPU data to GPU
    err = hipMemcpy(d_H, h_H, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        cout << "Copying host vectors into host memory unsuccessful." << endl;
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_G, h_H, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        cout << "Copying host vectors into host memory unsuccessful." << endl;
        exit(EXIT_FAILURE);
    }

    // (5) Kernel calculation
    int threadsPerBlock = 256;
    int blocksPerGrid = (totalElements + threadsPerBlock - 1)/threadsPerBlock;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Timing:
    hipEventRecord(start);

    // -> Main function
    for (int i = 0; i < I; i++)
    {
        UpdatingTemp<<<blocksPerGrid, threadsPerBlock>>>(d_H, d_G, plateSideLength, totalElements);
        copyArray <<<blocksPerGrid, threadsPerBlock>>>(d_H, d_G, totalElements);
        
	    err = hipGetLastError();
    	if (err != hipSuccess)
    	{
	    cout << "Kernel execusion failed" << endl;
            exit(EXIT_FAILURE);
    	}

    }
    hipEventRecord(stop);

    // (6) Send GPU data to CPU
    err = hipMemcpy(h_G, d_G, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
	    cout << "Unsuccessful copy to host device." << endl;
        exit(EXIT_FAILURE);
    }

    hipEventSynchronize(stop);

    // (7) Print time (in ms)
    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    if (err != hipSuccess)
    {
        exit(EXIT_FAILURE);
    }
    
    cout << fixed << showpoint;
    cout << setprecision(2);
    cout << time << endl;


    // (8) Save result to CSV file
    for(int i = 0; i < plateSideLength; ++i)
    {
        for(int j = 0; j < plateSideLength; ++j)
        {
            if (j != (plateSideLength - 1))
            {
                outputFile << to_string( h_G[i * plateSideLength + j] ) << ",";
            }
        }
        if (i != (plateSideLength - 1))
        {
            outputFile << "\n";
        }
    }

    // (9) Free GPU Memory
    err = hipFree(d_H);
    if (err != hipSuccess)
    {
        cout << "Free Memory fail." << endl;
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_G);
    if (err != hipSuccess)
    {
        cout << "Free Memory fail." << endl;
        exit(EXIT_FAILURE);
    }

    // (10) Free host memory
    free(h_H);
    free(h_G);

    outputFile.close();

    return 1;
}

/*
 * Function: bool is_numeric(const string& input, unsigned long& nInputNumber)
 *   Description : This is a function (provided by Dr. Hurley on Piazza) to check and see if the input is valid
 *   Return      : Return true if the input is valid; otherwise, return false.
 *   Parameters  : 
 *      const string& input: the command line argument we want to check for its validity
 *      unsigned long& nInputNumber: the variable storing the number that user enters in the commandline
 */
bool is_numeric(char* & input, int & nInputNumber)
{
    
    string s = input;

    // Check special case first
    if ( s.compare( "0.0" ) == 0 )
    {
        nInputNumber = 0;
        return true;
    }

    // If all letter is digit number, it is valid
    bool bRC = std::all_of(s.begin(), s.end(),                    // http://www.cplusplus.com/reference/algorithm/all_of/
        [](unsigned char c) { return ::isdigit(c); }              // https://www.geeksforgeeks.org/lambda-expression-in-c/
    );                                                            // http://www.cplusplus.com/reference/cctype/isdigit/
    
    if (bRC)
    {
        // Valid! Place the number to the desired variable
        nInputNumber = std::stoi(input);                         // https://www.cplusplus.com/reference/string/stoul/
        return true;
    }
    else
    {
        // There are some digit that is not a number
        return false;

    }
    
    return false;

}